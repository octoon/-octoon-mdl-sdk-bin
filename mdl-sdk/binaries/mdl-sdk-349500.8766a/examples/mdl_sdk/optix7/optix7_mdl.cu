#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2022 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

#include <optix.h>

#include "optix7_mdl.h"

#include <mi/neuraylib/target_code_types.h>


static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    RadiancePRD*           prd,
    RayFlags&              depth
#ifdef CONTRIB_IN_PAYLOAD
    , float3&              contrib
#endif
)
{
    uint32_t u0, u1;
    pack_pointer(prd, u0, u1);
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                     // rayTime
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        RAY_TYPE_RADIANCE,        // missSBTIndex
        u0, u1,
        reinterpret_cast<uint32_t&>(depth)
#ifdef CONTRIB_IN_PAYLOAD
        , reinterpret_cast<uint32_t&>(contrib.x),
        reinterpret_cast<uint32_t&>(contrib.y),
        reinterpret_cast<uint32_t&>(contrib.z)
#endif
    );
}


// simple Reinhard tonemapper + gamma + quantize
__forceinline__ __device__ uchar4 make_color(float3 const &c)
{
    const float burn_out = 0.1f;
    const float gamma = 2.2f;

    float3 val;
    val.x = c.x * (1.0f + c.x * burn_out) / (1.0f + c.x);
    val.y = c.y * (1.0f + c.y * burn_out) / (1.0f + c.y);
    val.z = c.z * (1.0f + c.z * burn_out) / (1.0f + c.z);

    return make_uchar4(
        static_cast<uint8_t>(powf(saturate(val.x), 1.0 / gamma)*255.0f),
        static_cast<uint8_t>(powf(saturate(val.y), 1.0 / gamma)*255.0f),
        static_cast<uint8_t>(powf(saturate(val.z), 1.0 / gamma)*255.0f),
        255u
    );
}


//------------------------------------------------------------------------------
//
// Ray generation function
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int    w   = params.width;
    const int    h   = params.height;
    const float3 eye = params.eye;
    const float3 U   = params.U;
    const float3 V   = params.V;
    const float3 W   = params.W;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;

    uint32_t seed = tea<4>(idx.y*w + idx.x, subframe_index);

    float3 result = make_float3(0.0f);
    int i = params.samples_per_launch;

    do
    {
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        const float2 d = 2.0f * make_float2(
                (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
                (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
            ) - 1.0f;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);
        float3 ray_origin    = eye;

        RadiancePRD prd;
#ifndef CONTRIB_IN_PAYLOAD
        prd.contribution = make_float3(0.f);
#endif
        prd.weight       = make_float3(1.f);
        prd.seed         = seed;
        prd.last_pdf     = -1.0f;

        RayFlags ray_flags = RAY_FLAGS_NONE;
        for (;; )
        {
            traceRadiance(
                params.handle,
                ray_origin,
                ray_direction,
                0.01f,  // tmin       // TODO: smarter offset
                1e16f,  // tmax
                &prd,
                ray_flags
#ifdef CONTRIB_IN_PAYLOAD
                , result
#endif
            );

            if ((ray_flags & RAY_FLAGS_DEPTH_MASK) >= MAX_DEPTH)
                break;

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            ray_flags = RayFlags(int(ray_flags) + 1);
        }

#ifndef CONTRIB_IN_PAYLOAD
        result += prd.contribution;
#endif
    } while (--i > 0);

    const uint3    launch_index = optixGetLaunchIndex();
    const uint32_t image_index  = launch_index.y * params.width + launch_index.x;
    float3         accum_color  = result / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = params.accum_buffer[image_index];
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = accum_color;
    if (params.frame_buffer)
        params.frame_buffer[image_index] = make_color(accum_color);
}


//------------------------------------------------------------------------------
//
// Miss function of radiance ray
//
//------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance()
{
    RadiancePRD* prd = get_radiance_prd();

    const float3 ray_dir = optixGetWorldRayDirection();

    float pdf;
    float3 radiance = environment_eval(pdf, ray_dir);

    // to incorporate the point light selection probability
    if (params.light.emission.x > 0.0f || params.light.emission.y > 0.0f || params.light.emission.z > 0.0f)
        pdf *= 0.5f;

    if (prd->last_pdf > 0.0f)
    {
        float mis_weight = prd->last_pdf / (prd->last_pdf + pdf);
        radiance *= mis_weight;
    }

#ifdef CONTRIB_IN_PAYLOAD
    set_radiance_payload_contrib(get_radiance_payload_contrib() + radiance * prd->weight);
#else
    prd->contribution += radiance * prd->weight;
#endif

    set_radiance_payload_depth(MAX_DEPTH);
}


//------------------------------------------------------------------------------
//
// Closest-hit function of occlusion ray
//
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__occlusion()
{
    set_occlusion_payload(true);
}
